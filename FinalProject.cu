#include "hip/hip_runtime.h"
/* This code will generate a Sobel image and a Gray Scale image. Uses OpenCV, to compile:
   nvcc FinalProject.cu `pkg-config --cflags --libs opencv`  

   Copyright (C) 2018  Jose Andres Cortez Villao

    This program is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.

    This program is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.

    You should have received a copy of the GNU General Public License
    along with this program.  If not, see <https://www.gnu.org/licenses/>.*/
	
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <opencv/highgui.h>
#include "utils/cheader.h"


typedef enum color {BLUE, GREEN, RED} Color;	//Constants that contains the values for each color of the image 

/*The gray function obtain an average of each pixel and assigned to the correct position in the array using 
Channels and step constants*/
__global__ void gray(unsigned char *src, unsigned char *dest, int width, int heigth, int step, int channels) { 
	int ren, col;
	float r, g, b;
	
	ren = blockIdx.x; // Variables that parallelize the code 
	col = threadIdx.x;
	r = 0; g = 0; b = 0;

	r += (float) src[(ren * step) + (col * channels) + RED];
	g += (float) src[(ren * step) + (col * channels) + GREEN];
	b += (float) src[(ren * step) + (col * channels) + BLUE];

	dest[(ren * step) + (col * channels) + RED] =  (unsigned char) ((r+g+b)/3);
	dest[(ren * step) + (col * channels) + GREEN] = (unsigned char) ((r+g+b)/3);
	dest[(ren * step) + (col * channels) + BLUE] = (unsigned char) ((r+g+b)/3);
}
/*The sobel function uses a convolution algorithm to obtain the edges of the image */

__global__ void sobel(unsigned char *src, unsigned char *dest, int width, int heigth, int step, int channels){
	int i, j;
	int ren, col, tmp_ren, tmp_col;
	int gx[3][3]={{-1,0,1},{-2,0,2},{-1,0,1}}; // gx is defined in the Sobel algorithm
	int gy[3][3]={{1,2,1},{0,0,0},{-1,-2,-1}}; // gy is defined in the Sobel algorithm 
	char temp[3][3];
	 
	ren = blockIdx.x;
	col = threadIdx.x;

	tmp_ren = 0;
	tmp_col = 0;
	
	//Multiplication of the 3x3 matrix for each color 
	for (i = -1; i < 2; i++) {
		for (j = -1; j < 2; j++) {
			temp[i+1][j+1]=(int) src[(ren * step) + (col * channels) + RED + i + 1];
			tmp_ren=tmp_ren + temp[i+1][j+1]*gx[i+1][j+1];
			tmp_col=tmp_col + temp[i+1][j+1]*gy[i+1][j+1];				
		}
	}
	dest[(ren * step) + (col * channels) + RED] =  (unsigned char) sqrtf(tmp_col*tmp_col+tmp_ren*tmp_ren);;

	tmp_ren = 0;
	tmp_col = 0;
	for (i = -1; i < 2; i++) {
		for (j = -1; j < 2; j++) {
			temp[i+1][j+1]=(int) src[(ren * step) + (col * channels) + GREEN + i + 1];
			tmp_ren=tmp_ren + temp[i+1][j+1]*gx[i+1][j+1];
			tmp_col=tmp_col + temp[i+1][j+1]*gy[i+1][j+1];				
		}
	}
	dest[(ren * step) + (col * channels) + GREEN] =  (unsigned char) sqrtf(tmp_col*tmp_col+tmp_ren*tmp_ren);;


	tmp_ren = 0;
	tmp_col = 0;
	for (i = -1; i < 2; i++) {
		for (j = -1; j < 2; j++) {
			temp[i+1][j+1]=(int) src[(ren * step) + (col * channels) + BLUE + i + 1];
			tmp_ren=tmp_ren + temp[i+1][j+1]*gx[i+1][j+1];
			tmp_col=tmp_col + temp[i+1][j+1]*gy[i+1][j+1];				
		}
	}
	dest[(ren * step) + (col * channels) + BLUE] =  (unsigned char) sqrtf(tmp_col*tmp_col+tmp_ren*tmp_ren);
}

int main(int argc, char* argv[]) {
	int i, step, size;
	double acum; 
	unsigned char *dev_src, *dev_gray,*dev_sobel;
		
	if (argc != 2) {
		printf("usage: %s source_file\n", argv[0]);
		return -1;
	}
	//Obtain and create the image using OpenCV 
	IplImage *src = cvLoadImage(argv[1], CV_LOAD_IMAGE_COLOR);
	IplImage *grayImage = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);
	IplImage *sobelImage = cvCreateImage(cvSize(src->width, src->height), IPL_DEPTH_8U, 3);
	
	//Check if there is an image as an argument
	if (!src) {
		printf("Could not load image file: %s\n", argv[1]);
		return -1;
	}
	
	size = src->width * src->height * src->nChannels * sizeof(uchar);
	hipMalloc((void**) &dev_src, size);
	hipMalloc((void**) &dev_gray, size);
	hipMalloc((void**) &dev_sobel, size);
	hipMemcpy(dev_src, src->imageData, size, hipMemcpyHostToDevice);
	
	acum = 0;
	step = src->widthStep / sizeof(uchar);
	
	//Compute the execution time for each function 
	printf("Starting...\n");
	for (i = 0; i < N; i++) {
		start_timer();
		gray<<<src->height, src->width>>>(dev_src, dev_gray, src->width, src->height, step, src->nChannels);
		hipMemcpy(grayImage->imageData, dev_gray, size, hipMemcpyDeviceToHost);
		acum += stop_timer();
	}
	
	for (i = 0; i < N; i++) {
		start_timer();
		hipMemcpy(dev_gray, grayImage->imageData, size, hipMemcpyHostToDevice);
		sobel<<<grayImage->height, grayImage->width>>>(dev_gray, dev_sobel, src->width-1, src->height-1, step, src->nChannels);
		hipMemcpy(sobelImage->imageData, dev_sobel, size, hipMemcpyDeviceToHost);
		acum += stop_timer();
	}
	//Free the memory of the GPU 
	hipFree(dev_gray);
	hipFree(dev_src);
	hipFree(dev_sobel);
	
	printf("avg time = %.5lf ms\n", (acum / (N*2)));
	
	cvShowImage("(Original)", src);
	cvShowImage("(Gray)", grayImage);
	cvShowImage("(Sobel)", sobelImage);
	cvWaitKey(0);
	cvWaitKey(0);
	cvDestroyWindow("Lenna (Original)");
	cvDestroyWindow("Lenna (Gray)");
	cvDestroyWindow("Lenna (Sobel)");

	return 0;
}
